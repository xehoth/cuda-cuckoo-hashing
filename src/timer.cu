#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/11.
//
#include <timer.cuh>
#include <numeric>
#include <iostream>

Timer::Timer() {
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
}

Timer::~Timer() {
  hipEventDestroy(start_event);
  hipEventDestroy(end_event);
}

void Timer::start() { hipEventRecord(start_event, nullptr); }

void Timer::end() {
  hipEventRecord(end_event, nullptr);
  hipEventSynchronize(end_event);
  float elapsed;
  hipEventElapsedTime(&elapsed, start_event, end_event);
  times.push_back(elapsed / 1000.0);
}

void Timer::clear() { times.clear(); }

double Timer::average() const {
  return std::accumulate(times.begin(), times.end(), 0.0) /
         static_cast<double>(times.size());
}

double Timer::stddev() const {
  double sum = std::accumulate(times.begin(), times.end(), 0.0);
  double avg = sum / static_cast<double>(times.size());
  double ret = 0;
  for (auto &v : times) ret += (v - avg) * (v - avg);
  ret = std::sqrt(ret / static_cast<double>(times.size()));
  return ret;
}

void Timer::report(std::uint32_t keys) const {
  double avg = average();
  double mkps = keys / 1e6 / avg;
  std::clog << mkps << "MTEPS, stddev: " << mkps * stddev() << ", time: " << avg * 1000
            << std::endl;
}
