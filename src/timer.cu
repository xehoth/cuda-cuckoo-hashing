#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/11.
//
#include <timer.cuh>
#include <numeric>
#include <iostream>

Timer::Timer() {
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
}

Timer::~Timer() {
  hipEventDestroy(start_event);
  hipEventDestroy(end_event);
}

void Timer::start() { hipEventRecord(start_event, nullptr); }

void Timer::end() {
  hipEventRecord(end_event, nullptr);
  hipEventSynchronize(end_event);
  float elapsed;
  hipEventElapsedTime(&elapsed, start_event, end_event);
  times.push_back(elapsed / 1000.0);
}

void Timer::clear() { times.clear(); }

double Timer::average() const {
  return std::accumulate(times.begin(), times.end(), 0.0) /
         static_cast<double>(times.size());
}

double Timer::stddev() const {
  double sum = std::accumulate(times.begin(), times.end(), 0.0);
  double avg = sum / static_cast<double>(times.size());
  double ret = 0;
  for (auto &v : times) ret += (v - avg) * (v - avg);
  ret = std::sqrt(ret / static_cast<double>(times.size()));
  return ret;
}

void Timer::report(std::uint32_t keys) const {
  double avg = average();
  double mops = keys / 1e6 / avg;
  std::cout << mops << "MOPS, stddev: " << 1000 * stddev()
            << ", time: " << avg * 1000 << std::endl;
}

std::string Timer::to_string(std::uint32_t keys) const {
  double avg = average();
  double mops = keys / 1e6 / avg;
  return std::to_string(mops) + " " + std::to_string(1000 * stddev()) + " " +
         std::to_string(avg * 1000) + '\n';
}
