#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/12.
//
#include <cstdint>
#include <hash_table.cuh>
#include <rng.cuh>
#include <timer.cuh>
#include <cstdio>
#include <fstream>

template <std::uint32_t s, std::uint32_t N_H>
std::string do_test1() {
  fprintf(stderr, "test1 (s = %u, t = %u):\n", s, N_H);
  constexpr std::uint32_t C = 1 << 25;
  constexpr std::uint32_t S = 1 << s;
  HashTable<C, 4 * s, N_H> table;
  fprintf(stderr, "  generate random set ... ");
  HostArray<std::uint32_t, S> h_set = generate_random_set<S>();
  fprintf(stderr, "done\n");
  DeviceArray<std::uint32_t, S> d_set;
  d_set = h_set;
  h_set.free();
  fprintf(stderr, "  begin testing ... \n");
  Timer timer;
  for (int i = 0; i < 5; ++i) {
    fprintf(stderr, "    round %d begin ... ", i);
    timer.start();
    table.insert(d_set);
    timer.end();
    fprintf(stderr, "done\n");
    table.clear();
  }
  fprintf(stderr, "  done\n");
  d_set.free();
  table.free();
  fprintf(stderr, "done\n\n");
  return timer.to_string(S);
}

void do_test1_all() {
  std::ofstream out("test1.bench");
  out << do_test1<10, 2>();
  out << do_test1<11, 2>();
  out << do_test1<12, 2>();
  out << do_test1<13, 2>();
  out << do_test1<14, 2>();
  out << do_test1<15, 2>();
  out << do_test1<16, 2>();
  out << do_test1<17, 2>();
  out << do_test1<18, 2>();
  out << do_test1<19, 2>();
  out << do_test1<20, 2>();
  out << do_test1<21, 2>();
  out << do_test1<22, 2>();
  out << do_test1<23, 2>();
  out << do_test1<24, 2>();
  out << do_test1<10, 3>();
  out << do_test1<11, 3>();
  out << do_test1<12, 3>();
  out << do_test1<13, 3>();
  out << do_test1<14, 3>();
  out << do_test1<15, 3>();
  out << do_test1<16, 3>();
  out << do_test1<17, 3>();
  out << do_test1<18, 3>();
  out << do_test1<19, 3>();
  out << do_test1<20, 3>();
  out << do_test1<21, 3>();
  out << do_test1<22, 3>();
  out << do_test1<23, 3>();
  out << do_test1<24, 3>();
}

int main() {
  do_test1_all();
  return 0;
}