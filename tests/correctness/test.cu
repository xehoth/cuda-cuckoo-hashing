//
// Created by xehoth on 2021/12/20.
//
#include "test.cuh"
#include <hash_table.cuh>
#include <rng.cuh>

void do_correctness_test() {
  printf("testing correctness ... ");
  constexpr std::uint32_t C = 1 << 25;
  constexpr std::uint32_t S = 1 << 24;
  HashTable<C, 4 * 24, 2> table;
  HostArray<std::uint32_t, S> h_set = generate_random_set<S>();
  HostArray<std::uint32_t, S> h_lookup_set = generate_lookup_set<S, 5>(h_set);
  DeviceArray<std::uint32_t, S> d_set;
  d_set = h_set;
  DeviceArray<std::uint32_t, S> d_lookup_set, d_res;
  HostArray<std::uint32_t, S> h_res;
  d_lookup_set = h_lookup_set;
  auto check_correctness = [&]() {
    std::unordered_set<std::uint32_t> hash_set;
    for (std::uint32_t i = 0; i < h_set.size(); ++i) hash_set.insert(h_set(i));
    HostArray<std::uint32_t, S> h_res;
    h_res = d_res;
    for (std::uint32_t i = 0; i < h_res.size(); ++i) {
      std::uint32_t expected = hash_set.count(h_lookup_set(i));
      if (h_res(i) != expected) {
        fprintf(stderr, "[wrong answer at (%u: %u), expected: %u]\n", i,
                h_lookup_set(i), expected);
        return;
      }
    }
    printf("[correct]\n");
  };
  Timer timer;
  table.insert_and_lookup(d_set, d_lookup_set, d_res, timer);
  checkCudaErrors(hipDeviceSynchronize());
  check_correctness();
  h_set.free();
  h_lookup_set.free();
  h_res.free();
  d_set.free();
  d_lookup_set.free();
  d_res.free();
  table.free();
}