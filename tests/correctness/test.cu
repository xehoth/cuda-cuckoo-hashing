//
// Created by xehoth on 2021/12/20.
//
#include "test.cuh"
#include <hash_table.cuh>
#include <rng.cuh>

void do_correctness_test() {
  RandomSetGenerator<(1 << 24)>::get();
  printf("testing correctness ... ");
  constexpr std::uint32_t C = 1 << 25;
  constexpr std::uint32_t S = 1 << 24;
  HashTable<C, 4 * 24, 2> table;
  HostArray<std::uint32_t, S> h_set, h_lookup_set;
  DeviceArray<std::uint32_t, S> d_set;
  RandomSetGenerator<S>::get()->generate_random_set(d_set);
  DeviceArray<std::uint32_t, S> d_lookup_set, d_res;
  RandomSetGenerator<S>::get()->generate_lookup_set<5>(d_lookup_set);
  HostArray<std::uint32_t, S> h_res;
  h_set = d_set;
  h_lookup_set = d_lookup_set;
  auto check_correctness = [&]() {
    std::unordered_set<std::uint32_t> hash_set;
    for (std::uint32_t i = 0; i < h_set.size(); ++i) hash_set.insert(h_set(i));
    HostArray<std::uint32_t, S> h_res;
    h_res = d_res;
    for (std::uint32_t i = 0; i < h_res.size(); ++i) {
      std::uint32_t expected = hash_set.count(h_lookup_set(i));
      if (h_res(i) != expected) {
        printf("[wrong answer at (%u: %u), expected: %u]\n", i, h_lookup_set(i),
               expected);
        return;
      }
    }
    printf("[correct]\n");
  };
  Timer timer;
  table.insert_and_lookup(d_set, d_lookup_set, d_res, timer);
  checkCudaErrors(hipDeviceSynchronize());
  check_correctness();
  h_set.free();
  h_lookup_set.free();
  h_res.free();
  d_set.free();
  d_lookup_set.free();
  d_res.free();
  table.free();
}