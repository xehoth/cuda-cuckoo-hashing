#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/12.
//
#include <hash_table.cuh>
#include <rng.cuh>
#include <fstream>

template <std::uint32_t C, std::uint32_t N_H>
std::string do_test3() {
  fprintf(stderr, "test3 (C = %u, t = %u):\n", C, N_H);
  HashTable<C, 4 * 24, N_H> table;
  constexpr std::uint32_t S = 1 << 24;
  fprintf(stderr, "  generate random set ... ");
  HostArray<std::uint32_t, S> h_set = generate_random_set<S>();
  fprintf(stderr, "done\n");
  DeviceArray<std::uint32_t, S> d_set;
  d_set = h_set;
  h_set.free();
  fprintf(stderr, "  begin testing ... \n");
  Timer timer;
  for (int i = 0; i < 5; ++i) {
    fprintf(stderr, "    round %d begin ... ", i);
    timer.start();
    table.insert(d_set);
    timer.end();
    fprintf(stderr, "done\n");
    table.clear();
  }
  fprintf(stderr, "  done\n");
  d_set.free();
  table.free();
  fprintf(stderr, "done\n\n");
  return timer.to_string(S);
}

void do_test3_all() {
  std::ofstream out("test3.bench");
  constexpr std::uint32_t S = 1 << 24;
#define _C(v) (static_cast<std::uint32_t>(S * (v) + 1 - 1e-10))
  out << do_test3<_C(1.1), 2>();
  out << do_test3<_C(1.2), 2>();
  out << do_test3<_C(1.3), 2>();
  out << do_test3<_C(1.4), 2>();
  out << do_test3<_C(1.5), 2>();
  out << do_test3<_C(1.6), 2>();
  out << do_test3<_C(1.7), 2>();
  out << do_test3<_C(1.8), 2>();
  out << do_test3<_C(1.9), 2>();
  out << do_test3<_C(2.0), 2>();
  out << do_test3<_C(1.01), 2>();
  out << do_test3<_C(1.02), 2>();
  out << do_test3<_C(1.05), 2>();
  out << do_test3<_C(1.1), 3>();
  out << do_test3<_C(1.2), 3>();
  out << do_test3<_C(1.3), 3>();
  out << do_test3<_C(1.4), 3>();
  out << do_test3<_C(1.5), 3>();
  out << do_test3<_C(1.6), 3>();
  out << do_test3<_C(1.7), 3>();
  out << do_test3<_C(1.8), 3>();
  out << do_test3<_C(1.9), 3>();
  out << do_test3<_C(2.0), 3>();
  out << do_test3<_C(1.01), 3>();
  out << do_test3<_C(1.02), 3>();
  out << do_test3<_C(1.05), 3>();
#undef _C
}

int main() {
  do_test3_all();
  return 0;
}