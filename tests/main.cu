//
// Created by xehoth on 2021/12/11.
//
//#include "insert/test.cuh"
//#include "lookup/test.cuh"
//#include "size/test.cuh"
//#include "bound/test.cuh"
//#include "correctness/test.cuh"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

int main() {
  freopen("test.log", "w", stderr);
#ifdef _WIN32
  system("main_test1");
  system("main_test2");
  system("main_test3");
  system("main_test4");
#else
  system("./main_test1");
  system("./main_test2");
  system("./main_test3");
  system("./main_test4");
#endif
  return 0;
}
