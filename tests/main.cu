#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/11.
//
#include "insert/test.cuh"
#include "lookup/test.cuh"
#include "size/test.cuh"
#include "bound/test.cuh"
#include "correctness/test.cuh"
#include <cstdio>

int main() {
  freopen("test.log", "w", stderr);
  do_correctness_test();
  do_test1_all();
  do_test2_all();
  do_test3_all();
  do_test4_all();
  return 0;
}
