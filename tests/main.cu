#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/11.
//
#include <timer.cuh>
#include <hash_table.cuh>
#include <set>
#include <random>
#include <iostream>
#include <rng.cuh>
__global__ void test() { printf("%d %d\n", blockIdx.x, threadIdx.x); }

int main() {
  HashTable<std::uint32_t((1 << 24) * 1.01), 3> table;
  constexpr int S = 1 << 24;
  HostArray<std::uint32_t, S> h;
  std::set<std::uint32_t> set;
  std::mt19937 engine;
  std::uniform_int_distribution<std::uint32_t> dis;
  //  for (int i = 0; i < S; ++i) {
  //    std::uint32_t x;
  //    do {
  //      x = dis(engine);
  //    } while (set.count(x) || x == -1u);
  //    set.insert(x);
  //  }
  //  auto it = set.begin();
  std::cerr << "generate begin" << std::endl;
#pragma omp parallel for
  for (std::uint32_t i = 0; i < h.size(); ++i) {
    h(i) = dis(engine);
    while (h(i) == -1u) h(i) = dis(engine);
  }
  std::cerr << "generate done" << std::endl;
  DeviceArray<std::uint32_t, S> d, lookup, res;
  d = h;
  lookup = h;
  std::uint32_t a = 1000;
  hipMemcpy(lookup.data, &a, sizeof(std::uint32_t), hipMemcpyHostToDevice);
  res.constant_fill<0>();

  Timer timer;
  for (int i = 0; i < 5; ++i) {
    timer.start();
    table.insert(d);
    timer.end();
//    table.insert_and_lookup<S, S>(d, lookup, res, timer);
    table.clear();
  }
//  HostArray<std::uint32_t, S> h_res;
//  h_res = res;
//  for (std::uint32_t i = 0; i < h_res.size(); ++i) {
//    if (h_res(i) != 1) {
//      printf("wrong answer %d !!!!", i);
//    }
//  }
  table.print();
  hipDeviceSynchronize();
  table.free();
  d.free();
  h.free();
  timer.report(S);
  return 0;
}