#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/11.
//
#include <timer.cuh>
#include <hash_table.cuh>
#include <set>
#include <random>
#include <iostream>

__global__ void test() { printf("%d %d\n", blockIdx.x, threadIdx.x); }

int main() {
  HashTable<std::uint32_t((1 << 24) * 1.02), 3> table;
  constexpr int S = 1 << 24;
  HostArray<std::uint32_t, S> h;
  std::set<std::uint32_t> set;
  std::mt19937 engine;
  std::uniform_int_distribution<std::uint32_t> dis;
  //  for (int i = 0; i < S; ++i) {
  //    std::uint32_t x;
  //    do {
  //      x = dis(engine);
  //    } while (set.count(x) || x == -1u);
  //    set.insert(x);
  //  }
  //  auto it = set.begin();
  std::cerr << "generate begin" << std::endl;
#pragma omp parallel for
  for (std::uint32_t i = 0; i < h.size(); ++i) {
    h(i) = dis(engine);
    while (h(i) == -1u) h(i) = dis(engine);
  }
  std::cerr << "generate done" << std::endl;
  DeviceArray<std::uint32_t, S> d;
  d = h;

  Timer timer;
  for (int i = 0; i < 5; ++i) {
    timer.start();
    table.insert(d);
    timer.end();
    table.clear();
  }
  table.print();
  hipDeviceSynchronize();
  table.free();
  d.free();
  h.free();
  timer.report(1 << 25);
  return 0;
}