#include "hip/hip_runtime.h"
//
// Created by xehoth on 2021/12/12.
//
#include <rng.cuh>
#include <hash_table.cuh>
#include <cstdint>
#include <fstream>

template <std::uint32_t l, std::uint32_t N_H>
std::string do_test4() {
  fprintf(stderr, "test3 (C = %.1f, t = %u):\n", l / 10.0, N_H);
  constexpr std::uint32_t S = 1 << 24;
  constexpr auto C = static_cast<std::uint32_t>(S * 1.4 + 1 - 1e-10);
  constexpr auto bound = static_cast<std::uint32_t>(24 * l / 10.0 + 1 - 1e-10);
  HashTable<C, bound, N_H> table;
  fprintf(stderr, "  generate random set ... ");
  HostArray<std::uint32_t, S> h_set = generate_random_set<S>();
  fprintf(stderr, "done\n");
  DeviceArray<std::uint32_t, S> d_set;
  d_set = h_set;
  h_set.free();
  fprintf(stderr, "  begin testing ... \n");
  Timer timer;
  for (int i = 0; i < 5; ++i) {
    fprintf(stderr, "    round %d begin ... ", i);
    timer.start();
    table.insert(d_set);
    timer.end();
    fprintf(stderr, "done\n");
    table.clear();
  }
  fprintf(stderr, "  done\n");
  d_set.free();
  table.free();
  fprintf(stderr, "done\n\n");
  return timer.to_string(S);
}

void do_test4_all() {
  std::ofstream out("test4.bench");
  out << do_test4<2, 2>();
  out << do_test4<4, 2>();
  out << do_test4<6, 2>();
  out << do_test4<8, 2>();
  out << do_test4<10, 2>();
  out << do_test4<12, 2>();
  out << do_test4<14, 2>();
  out << do_test4<16, 2>();
  out << do_test4<18, 2>();
  out << do_test4<20, 2>();
  out << do_test4<30, 2>();
  out << do_test4<40, 2>();
  out << do_test4<60, 2>();
  out << do_test4<80, 2>();
  out << do_test4<120, 2>();
  out << do_test4<160, 2>();
  out << do_test4<2, 3>();
  out << do_test4<4, 3>();
  out << do_test4<6, 3>();
  out << do_test4<8, 3>();
  out << do_test4<10, 3>();
  out << do_test4<12, 3>();
  out << do_test4<14, 3>();
  out << do_test4<16, 3>();
  out << do_test4<18, 3>();
  out << do_test4<20, 3>();
  out << do_test4<30, 3>();
  out << do_test4<40, 3>();
  out << do_test4<60, 3>();
  out << do_test4<80, 3>();
  out << do_test4<120, 3>();
  out << do_test4<160, 3>();
}

int main() {
  do_test4_all();
  return 0;
}